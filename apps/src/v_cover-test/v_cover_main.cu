#include "v_cover.h"
#include "v_cover.cu"
#include <fstream>

v_coverMacro;
#include "graph.h"
#include "verify_v_cover.h"
int v_cover_CPU(int* G0, int* G1, bool * selectEdge) {

    {
        remain = (int)(NumEdges * 2);
        h___S4 = 0;
        err = hipMemcpyToSymbol(HIP_SYMBOL(__S4), &h___S4, sizeof(int), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;
        hipOccupancyMaxPotentialBlockSize(&gm_minGridSize, &gm_blockSize,forEachKernel0, 0, 0);
        gm_gridSize = (NumNodes + 1 + gm_blockSize - 1) / gm_blockSize;
        gm_numBlocksStillToProcess = gm_gridSize, gm_offsetIntoBlocks = 0;
        while (gm_numBlocksStillToProcess > 0) {
            if (gm_numBlocksStillToProcess > gm_minGridSize)
                gm_numBlocksKernelParameter = gm_minGridSize;
            else
                gm_numBlocksKernelParameter = gm_numBlocksStillToProcess;
            forEachKernel0<<<gm_numBlocksKernelParameter, gm_blockSize>>>(G0, G1, NumNodes, NumEdges, selectEdge, Deg, Covered, gm_offsetIntoBlocks);
            CUDA_ERR_CHECK;
            gm_numBlocksStillToProcess -= gm_minGridSize;
            gm_offsetIntoBlocks += gm_minGridSize * gm_blockSize;
        }
        hipOccupancyMaxPotentialBlockSize(&gm_minGridSize, &gm_blockSize,forEachKernel1, 0, 0);
        gm_gridSize = (NumEdges + gm_blockSize - 1) / gm_blockSize;
        gm_numBlocksStillToProcess = gm_gridSize, gm_offsetIntoBlocks = 0;
        while (gm_numBlocksStillToProcess > 0) {
            if (gm_numBlocksStillToProcess > gm_minGridSize)
                gm_numBlocksKernelParameter = gm_minGridSize;
            else
                gm_numBlocksKernelParameter = gm_numBlocksStillToProcess;
            forEachKernel1<<<gm_numBlocksKernelParameter, gm_blockSize>>>(G0, G1, NumNodes, NumEdges, selectEdge, gm_offsetIntoBlocks);
            CUDA_ERR_CHECK;
            gm_numBlocksStillToProcess -= gm_minGridSize;
            gm_offsetIntoBlocks += gm_minGridSize * gm_blockSize;
        }
        while (remain > 0)
        {
            h_max_val = 0;
            err = hipMemcpyToSymbol(HIP_SYMBOL(max_val), &h_max_val, sizeof(int), 0, hipMemcpyHostToDevice);
            CUDA_ERR_CHECK;
            hipOccupancyMaxPotentialBlockSize(&gm_minGridSize, &gm_blockSize,forEachKernel2, 0, 0);
            gm_gridSize = (NumNodes + 1 + gm_blockSize - 1) / gm_blockSize;
            gm_numBlocksStillToProcess = gm_gridSize, gm_offsetIntoBlocks = 0;
            while (gm_numBlocksStillToProcess > 0) {
                if (gm_numBlocksStillToProcess > gm_minGridSize)
                    gm_numBlocksKernelParameter = gm_minGridSize;
                else
                    gm_numBlocksKernelParameter = gm_numBlocksStillToProcess;
                forEachKernel2<<<gm_numBlocksKernelParameter, gm_blockSize>>>(G0, G1, NumNodes, NumEdges, selectEdge, Covered, Deg, gm_offsetIntoBlocks);
                CUDA_ERR_CHECK;
                gm_numBlocksStillToProcess -= gm_minGridSize;
                gm_offsetIntoBlocks += gm_minGridSize * gm_blockSize;
            }
            bool tempVar0 = true;
            err = hipMemcpyFromSymbol(&h_from, HIP_SYMBOL(from), sizeof(int), 0, hipMemcpyDeviceToHost);
            CUDA_ERR_CHECK;
            err = hipMemcpy(Covered + h_from, &tempVar0, 1 * sizeof(bool), hipMemcpyHostToDevice);
            CUDA_ERR_CHECK;
            tempVar0 = true;
            err = hipMemcpyFromSymbol(&h_to, HIP_SYMBOL(to), sizeof(int), 0, hipMemcpyDeviceToHost);
            CUDA_ERR_CHECK;
            err = hipMemcpy(Covered + h_to, &tempVar0, 1 * sizeof(bool), hipMemcpyHostToDevice);
            CUDA_ERR_CHECK;
            tempVar0 = true;
            err = hipMemcpyFromSymbol(&h_e, HIP_SYMBOL(e), sizeof(int), 0, hipMemcpyDeviceToHost);
            CUDA_ERR_CHECK;
            err = hipMemcpy(selectEdge + h_e, &tempVar0, 1 * sizeof(bool), hipMemcpyHostToDevice);
            CUDA_ERR_CHECK;
            int tempVar1 = 0;
            err = hipMemcpyFromSymbol(&h_from, HIP_SYMBOL(from), sizeof(int), 0, hipMemcpyDeviceToHost);
            CUDA_ERR_CHECK;
            err = hipMemcpy(Deg + h_from, &tempVar1, 1 * sizeof(int), hipMemcpyHostToDevice);
            CUDA_ERR_CHECK;
            tempVar1 = 0;
            err = hipMemcpyFromSymbol(&h_to, HIP_SYMBOL(to), sizeof(int), 0, hipMemcpyDeviceToHost);
            CUDA_ERR_CHECK;
            err = hipMemcpy(Deg + h_to, &tempVar1, 1 * sizeof(int), hipMemcpyHostToDevice);
            CUDA_ERR_CHECK;
            err = hipMemcpyFromSymbol(&h_max_val, HIP_SYMBOL(max_val), sizeof(int), 0, hipMemcpyDeviceToHost);
            CUDA_ERR_CHECK;
            remain = remain - h_max_val;
        }
        hipOccupancyMaxPotentialBlockSize(&gm_minGridSize, &gm_blockSize,forEachKernel3, 0, 0);
        gm_gridSize = (NumNodes + 1 + gm_blockSize - 1) / gm_blockSize;
        gm_numBlocksStillToProcess = gm_gridSize, gm_offsetIntoBlocks = 0;
        while (gm_numBlocksStillToProcess > 0) {
            if (gm_numBlocksStillToProcess > gm_minGridSize)
                gm_numBlocksKernelParameter = gm_minGridSize;
            else
                gm_numBlocksKernelParameter = gm_numBlocksStillToProcess;
            forEachKernel3<<<gm_numBlocksKernelParameter, gm_blockSize>>>(G0, G1, NumNodes, NumEdges, selectEdge, Covered, gm_offsetIntoBlocks);
            CUDA_ERR_CHECK;
            gm_numBlocksStillToProcess -= gm_minGridSize;
            gm_offsetIntoBlocks += gm_minGridSize * gm_blockSize;
        }
        err = hipMemcpyFromSymbol(&h___S4, HIP_SYMBOL(__S4), sizeof(int), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        return h___S4;
    }
}


using namespace std;
// v_cover -? : for how to run generated main program
int main(int argc, char* argv[])
{

    if (argc != 2 || argv[1] == NULL) {
        printf("Wrong Number of Arguments");
        exit(1);
    }
    ifstream inputFile;
    inputFile.open(argv[1]);
    if (!inputFile.is_open()){
        printf("invalid file");
        exit(1);
    }
    inputFile >> NumNodes >> NumEdges;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    err = hipMalloc((void **)&G0, (NumNodes + 2) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&G1, (NumEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&selectEdge, (NumEdges + 1) * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&Deg, (NumNodes + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&Covered, (NumNodes + 1) * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&host_threadBlockBarrierReached, 10000 * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMemset(host_threadBlockBarrierReached, 0x0, 10000 * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMemcpyToSymbol(HIP_SYMBOL(gm_threadBlockBarrierReached), &host_threadBlockBarrierReached, sizeof(bool *), 0, hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    int* h_G[2];
    printf("Graph Population began\n");
    populate(argv[1], h_G);
    printf("Graph Population end\n");
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Loading time(milliseconds)  = %f\n", elapsedTime);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    int MainReturn;
    MainReturn = v_cover_CPU(G0, G1, selectEdge);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Execution time(milliseconds)  = %f\n", elapsedTime);
    bool gm_verify = verifyv_cover(h_G);
    if (!gm_verify) {
        printf("Verification Failed\n");
        return -1;
    } else {
        printf("Verification Success\n");
    }
    err = hipFree(G0);
    CUDA_ERR_CHECK;
    err = hipFree(G1);
    CUDA_ERR_CHECK;
    err = hipFree(selectEdge);
    CUDA_ERR_CHECK;
    err = hipFree(Deg);
    CUDA_ERR_CHECK;
    err = hipFree(Covered);
    CUDA_ERR_CHECK;
    err = hipFree(host_threadBlockBarrierReached);
    CUDA_ERR_CHECK;
    free(h_G[0]);
    free(h_G[1]);
    printf("Return value = %d\n", MainReturn);return MainReturn;
}
