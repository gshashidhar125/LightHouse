#include "triangle_counting.h"
#include "triangle_counting.cu"
#include <fstream>

triangle_countingMacro;
#include "graph.h"
#include "verify_triangle_counting.h"
unsigned long long int triangle_counting_CPU(int* G0, int* G1) {

    {
        h_T = 0;
        err = hipMemcpyToSymbol(HIP_SYMBOL(T), &h_T, sizeof(unsigned long long int), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;
        hipOccupancyMaxPotentialBlockSize(&gm_minGridSize, &gm_blockSize,forEachKernel0, 0, 0);
        gm_gridSize = (NumNodes + 1 + gm_blockSize - 1) / gm_blockSize;
        gm_numBlocksStillToProcess = gm_gridSize, gm_offsetIntoBlocks = 0;
        while (gm_numBlocksStillToProcess > 0) {
            if (gm_numBlocksStillToProcess > gm_minGridSize)
                gm_numBlocksKernelParameter = gm_minGridSize;
            else
                gm_numBlocksKernelParameter = gm_numBlocksStillToProcess;
            forEachKernel0<<<gm_numBlocksKernelParameter, gm_blockSize>>>(G0, G1, NumNodes, NumEdges, gm_offsetIntoBlocks, h_inNode);
            CUDA_ERR_CHECK;
            gm_numBlocksStillToProcess -= gm_minGridSize;
            gm_offsetIntoBlocks += gm_minGridSize * gm_blockSize;
        }
        err = hipMemcpyFromSymbol(&h_T, HIP_SYMBOL(T), sizeof(unsigned long long int), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        return h_T;
    }
}


using namespace std;
// triangle_counting -? : for how to run generated main program
int main(int argc, char* argv[])
{

    if (argc != 2 || argv[1] == NULL) {
        printf("Wrong Number of Arguments");
        exit(1);
    }
    ifstream inputFile;
    inputFile.open(argv[1]);
    if (!inputFile.is_open()){
        printf("invalid file");
        exit(1);
    }
    inputFile >> NumNodes >> NumEdges;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    err = hipMalloc((void **)&G0, (NumNodes + 2) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&G1, (NumEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&h_inNode, (NumEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&host_threadBlockBarrierReached, 10000 * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMemset(host_threadBlockBarrierReached, 0x0, 10000 * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMemcpyToSymbol(HIP_SYMBOL(gm_threadBlockBarrierReached), &host_threadBlockBarrierReached, sizeof(bool *), 0, hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    int* h_G[2];
    printf("Graph Population began\n");
    populate(argv[1], h_G);
    printf("Graph Population end\n");
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Loading time(milliseconds)  = %f\n", elapsedTime);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    unsigned long long int MainReturn;
    MainReturn = triangle_counting_CPU(G0, G1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Execution time(milliseconds)  = %f\n", elapsedTime);
    bool gm_verify = verifytriangle_counting(h_G);
    if (!gm_verify) {
        printf("Verification Failed\n");
        return -1;
    } else {
        printf("Verification Success\n");
    }
    err = hipFree(G0);
    CUDA_ERR_CHECK;
    err = hipFree(G1);
    CUDA_ERR_CHECK;
    err = hipFree(h_inNode);
    CUDA_ERR_CHECK;
    err = hipFree(host_threadBlockBarrierReached);
    CUDA_ERR_CHECK;
    free(h_G[0]);
    free(h_G[1]);
    printf("Return value = %d\n", MainReturn);return MainReturn;
}
