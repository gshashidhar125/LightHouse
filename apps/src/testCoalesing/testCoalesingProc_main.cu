#include "testCoalesingProc.h"
#include "testCoalesingProc.cu"
#include <fstream>

testCoalescingMacro;
#include "graph.h"
#include "verify_testCoalescing.h"
void testCoalescing_CPU(int* G0, int* G1, int * A) {

    {
        hipOccupancyMaxPotentialBlockSize(&gm_minGridSize, &gm_blockSize,forEachKernel0, 0, 0);
        gm_gridSize = (NumEdges + gm_blockSize - 1) / gm_blockSize;
        gm_numBlocksStillToProcess = gm_gridSize, gm_offsetIntoBlocks = 0;
        while (gm_numBlocksStillToProcess > 0) {
            if (gm_numBlocksStillToProcess > gm_minGridSize)
                gm_numBlocksKernelParameter = gm_minGridSize;
            else
                gm_numBlocksKernelParameter = gm_numBlocksStillToProcess;
            forEachKernel0<<<gm_numBlocksKernelParameter, gm_blockSize>>>(G0, G1, NumNodes, NumEdges, A, gm_offsetIntoBlocks);
            CUDA_ERR_CHECK;
            gm_numBlocksStillToProcess -= gm_minGridSize;
            gm_offsetIntoBlocks += gm_minGridSize * gm_blockSize;
        }
    }
}


using namespace std;
// testCoalesingProc -? : for how to run generated main program
int main(int argc, char* argv[])
{

    if (argc != 2 || argv[1] == NULL) {
        printf("Wrong Number of Arguments");
        exit(1);
    }
    ifstream inputFile;
    inputFile.open(argv[1]);
    if (!inputFile.is_open()){
        printf("invalid file");
        exit(1);
    }
    inputFile >> NumNodes >> NumEdges;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    err = hipMalloc((void **)&G0, (NumNodes + 2) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&G1, (NumEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&A, (NumEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&host_threadBlockBarrierReached, 10000 * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMemset(host_threadBlockBarrierReached, 0x0, 10000 * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMemcpyToSymbol(HIP_SYMBOL(gm_threadBlockBarrierReached), &host_threadBlockBarrierReached, sizeof(bool *), 0, hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    int* h_G[2];
    printf("Graph Population began\n");
    populate(argv[1], h_G);
    printf("Graph Population end\n");
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Loading time(milliseconds)  = %f\n", elapsedTime);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    testCoalescing_CPU(G0, G1, A);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Execution time(milliseconds)  = %f\n", elapsedTime);
    bool gm_verify = verifytestCoalescing(h_G);
    if (!gm_verify) {
        printf("Verification Failed\n");
        return -1;
    } else {
        printf("Verification Success\n");
    }
    err = hipFree(G0);
    CUDA_ERR_CHECK;
    err = hipFree(G1);
    CUDA_ERR_CHECK;
    err = hipFree(A);
    CUDA_ERR_CHECK;
    err = hipFree(host_threadBlockBarrierReached);
    CUDA_ERR_CHECK;
    free(h_G[0]);
    free(h_G[1]);
}
