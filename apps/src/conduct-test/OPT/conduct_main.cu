#include "conduct.h"
#include "conduct.cu"
#include <fstream>

conductMacro;
#include "graph.h"
#include "verify_conduct.h"
float conduct_CPU(int* G0, int* G1, int * member, int num) {

    {
        h___S2 = 0;
        err = hipMemcpyToSymbol(HIP_SYMBOL(__S2), &h___S2, sizeof(int), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;
        h___S3 = 0;
        err = hipMemcpyToSymbol(HIP_SYMBOL(__S3), &h___S3, sizeof(int), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;
        h___S4 = 0;
        err = hipMemcpyToSymbol(HIP_SYMBOL(__S4), &h___S4, sizeof(int), 0, hipMemcpyHostToDevice);
        CUDA_ERR_CHECK;
        hipOccupancyMaxPotentialBlockSize(&gm_minGridSize, &gm_blockSize,forEachKernel0, 0, 0);
        gm_gridSize = (NumNodes + 1 + gm_blockSize - 1) / gm_blockSize;
        gm_numBlocksStillToProcess = gm_gridSize, gm_offsetIntoBlocks = 0;
        while (gm_numBlocksStillToProcess > 0) {
            if (gm_numBlocksStillToProcess > gm_minGridSize)
                gm_numBlocksKernelParameter = gm_minGridSize;
            else
                gm_numBlocksKernelParameter = gm_numBlocksStillToProcess;
            forEachKernel0<<<gm_numBlocksKernelParameter, gm_blockSize>>>(G0, G1, NumNodes, NumEdges, edgeFrom, member, num, gm_offsetIntoBlocks);
            CUDA_ERR_CHECK;
            gm_numBlocksStillToProcess -= gm_minGridSize;
            gm_offsetIntoBlocks += gm_minGridSize * gm_blockSize;
        }
        hipOccupancyMaxPotentialBlockSize(&gm_minGridSize, &gm_blockSize,forEachKernel1, 0, 0);
        gm_gridSize = (NumNodes + 1 + gm_blockSize - 1) / gm_blockSize;
        gm_numBlocksStillToProcess = gm_gridSize, gm_offsetIntoBlocks = 0;
        while (gm_numBlocksStillToProcess > 0) {
            if (gm_numBlocksStillToProcess > gm_minGridSize)
                gm_numBlocksKernelParameter = gm_minGridSize;
            else
                gm_numBlocksKernelParameter = gm_numBlocksStillToProcess;
            forEachKernel1<<<gm_numBlocksKernelParameter, gm_blockSize>>>(G0, G1, NumNodes, NumEdges, edgeFrom, member, num, gm_offsetIntoBlocks);
            CUDA_ERR_CHECK;
            gm_numBlocksStillToProcess -= gm_minGridSize;
            gm_offsetIntoBlocks += gm_minGridSize * gm_blockSize;
        }
        hipOccupancyMaxPotentialBlockSize(&gm_minGridSize, &gm_blockSize,forEachKernel2, 0, 0);
        gm_gridSize = (NumEdges + gm_blockSize - 1) / gm_blockSize;
        gm_numBlocksStillToProcess = gm_gridSize, gm_offsetIntoBlocks = 0;
        while (gm_numBlocksStillToProcess > 0) {
            if (gm_numBlocksStillToProcess > gm_minGridSize)
                gm_numBlocksKernelParameter = gm_minGridSize;
            else
                gm_numBlocksKernelParameter = gm_numBlocksStillToProcess;
            forEachKernel2<<<gm_numBlocksKernelParameter, gm_blockSize>>>(G0, G1, NumNodes, NumEdges, edgeFrom, member, num, gm_offsetIntoBlocks);
            CUDA_ERR_CHECK;
            gm_numBlocksStillToProcess -= gm_minGridSize;
            gm_offsetIntoBlocks += gm_minGridSize * gm_blockSize;
        }
        err = hipMemcpyFromSymbol(&h___S2, HIP_SYMBOL(__S2), sizeof(int), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        err = hipMemcpyFromSymbol(&h___S3, HIP_SYMBOL(__S3), sizeof(int), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        err = hipMemcpyFromSymbol(&h___S2, HIP_SYMBOL(__S2), sizeof(int), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        err = hipMemcpyFromSymbol(&h___S3, HIP_SYMBOL(__S3), sizeof(int), 0, hipMemcpyDeviceToHost);
        CUDA_ERR_CHECK;
        m = (float)((h___S2 < h___S3)?h___S2:h___S3);
        if (m == 0) {
            err = hipMemcpyFromSymbol(&h___S4, HIP_SYMBOL(__S4), sizeof(int), 0, hipMemcpyDeviceToHost);
            CUDA_ERR_CHECK;
            return (h___S4 == 0)?((float)(0.000000)):FLT_MAX;
        }
        else {
            err = hipMemcpyFromSymbol(&h___S4, HIP_SYMBOL(__S4), sizeof(int), 0, hipMemcpyDeviceToHost);
            CUDA_ERR_CHECK;
            return ((float)h___S4) / m;
        }
    }
}


using namespace std;
// conduct -? : for how to run generated main program
int main(int argc, char* argv[])
{

    if (argc != 2 || argv[1] == NULL) {
        printf("Wrong Number of Arguments");
        exit(1);
    }
    ifstream inputFile;
    inputFile.open(argv[1]);
    if (!inputFile.is_open()){
        printf("invalid file");
        exit(1);
    }
    inputFile >> NumNodes >> NumEdges;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    err = hipMalloc((void **)&G0, (NumNodes + 2) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&G1, (NumEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&member, (NumNodes + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&edgeFrom, (NumEdges + 1) * sizeof(int));
    CUDA_ERR_CHECK;
    err = hipMalloc((void **)&host_threadBlockBarrierReached, 10000 * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMemset(host_threadBlockBarrierReached, 0x0, 10000 * sizeof(bool));
    CUDA_ERR_CHECK;
    err = hipMemcpyToSymbol(HIP_SYMBOL(gm_threadBlockBarrierReached), &host_threadBlockBarrierReached, sizeof(bool *), 0, hipMemcpyHostToDevice);
    CUDA_ERR_CHECK;
    int* h_G[2];
    printf("Graph Population began\n");
    populate(argv[1], h_G);
    printf("Graph Population end\n");
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Loading time(milliseconds)  = %f\n", elapsedTime);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    float MainReturn;
    MainReturn = conduct_CPU(G0, G1, member, num);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Execution time(milliseconds)  = %f\n", elapsedTime);
    bool gm_verify = verifyconduct(h_G);
    if (!gm_verify) {
        printf("Verification Failed\n");
        return -1;
    } else {
        printf("Verification Success\n");
    }
    err = hipFree(G0);
    CUDA_ERR_CHECK;
    err = hipFree(G1);
    CUDA_ERR_CHECK;
    err = hipFree(member);
    CUDA_ERR_CHECK;
    err = hipFree(host_threadBlockBarrierReached);
    CUDA_ERR_CHECK;
    free(h_G[0]);
    free(h_G[1]);
    printf("Return value = %f\n", MainReturn);return MainReturn;
}
