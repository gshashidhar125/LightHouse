#include "hip/hip_runtime.h"

hop_distMacroGPU;
#include "GlobalBarrier.cuh"
__global__ void forEachKernel0 (int *G0, int *G1, int NumNodes, int NumEdges, int * dist, int root, bool * updated, int * dist_nxt, bool * updated_nxt, int gm_offsetIntoBlocks) {
    kernelMacro0;
    int tId = blockIdx.x * blockDim.x + threadIdx.x + gm_offsetIntoBlocks;
    if (tId >= NumNodes + 1) {
        return;
    }
    t0 = tId;
    {
        dist[t0] = (t0 == root)?0:2147483647;
        updated[t0] = (t0 == root)?true:false;
        dist_nxt[t0] = dist[t0];
        updated_nxt[t0] = updated[t0];
    }
}
__global__ void forEachKernel1 (int *G0, int *G1, int NumNodes, int NumEdges, int * dist, int root, bool * updated, int * dist_nxt, bool * updated_nxt, int gm_offsetIntoBlocks) {
    kernelMacro1;
    int tId = blockIdx.x * blockDim.x + threadIdx.x + gm_offsetIntoBlocks;
    if (tId >= NumNodes + 1) {
        return;
    }
    n = tId;
    {
        if (updated[n])
        {
            for (int iter = G0[n], s = G1[iter]; iter < G0[n + 1]; iter++, s = G1[iter]) {
                localExpr = dist_nxt[s];
                expr = dist[n] + 1;
                atomicMin(&dist_nxt[s], expr);
                if (localExpr > expr) {
                    updated_nxt[s] = true;
                }
            }
        }
    }
}
__global__ void forEachKernel2 (int *G0, int *G1, int NumNodes, int NumEdges, int * dist, int root, int * dist_nxt, bool * updated, bool * updated_nxt, int gm_offsetIntoBlocks) {
    kernelMacro2;
    int tId = blockIdx.x * blockDim.x + threadIdx.x + gm_offsetIntoBlocks;
    if (tId >= NumNodes + 1) {
        return;
    }
    t4 = tId;
    {
        dist[t4] = dist_nxt[t4];
        updated[t4] = updated_nxt[t4];
        updated_nxt[t4] = false;
        atomicOr((int *)&__E8, (int)updated[t4]);
    }
}
